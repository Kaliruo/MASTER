#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>


__global__ void kreduce(unsigned int *d_vec, int n){
	unsigned int tid= threadIdx.x;
	unsigned int offset;
	printf("oui%u\n",tid);
	for(offset=1; offset<= n/2; offset*=2){
		printf("oui%u\n",offset);
		if((tid%(2*offset))==0 && tid<n){
			d_vec[tid]+=d_vec[tid+offset];}
		__syncthreads();}
	}


void reduce(unsigned int *vec, unsigned int *sum, int size){
	unsigned int *d_vec;
	int bytes = size*sizeof(unsigned int);
	printf("oui%i\n",bytes);
	hipMalloc((void**)&d_vec, bytes);
	hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);
	
	kreduce<<<1,size>>>(d_vec, size);

	hipMemcpy(sum, d_vec, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(d_vec);
	}

int main(int argc, char **argv){
  if (argc < 2){
    printf("Usage: <filename>\n");
    exit(-1);
  }
unsigned int* vec;
int size;
FILE *f = fopen(argv[1],"r");
fscanf(f,"%d\n",&size);
size = 1 << size;
  if (size >= (1 << 20)){
    printf("Size (%u) is too large: size is limited to 2^20\n",size);
    exit(-1);
  }
vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
for (int i=0; i<size; i++){
    fscanf(f, "%u\n",&(vec[i]));
  }
unsigned int sum =0;

reduce(vec, &sum, size);
printf("sum=%u\n", sum);
return 0;

}

