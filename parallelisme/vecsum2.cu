#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>


__global__ void kreduce(unsigned int *d_vec, int n){
	unsigned int index= blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int offset;
	for(offset=1; offset<= n/2; offset*=2){
		if((index%(2*offset))==0 && index<n){
			d_vec[index]+=d_vec[index+offset];}
		__syncthreads();}
	}


void reduce(unsigned int *vec, unsigned int *sum, int size){
	unsigned int *d_vec;
	int bytes = size*sizeof(unsigned int);
	int gsize=((size+1024-1)/1024);
	printf("gsize=%i\n", gsize);
	hipMalloc((void**)&d_vec, bytes);
	hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);

	kreduce<<<gsize,1024>>>(d_vec, size);
	kreduce<<<1,1024>>>(d_vec, size);

	hipMemcpy(sum, d_vec, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(d_vec);
	}

int main(int argc, char **argv){
  if (argc < 2){
    printf("Usage: <filename>\n");
    exit(-1);
  }
unsigned int* vec;
int size;
FILE *f = fopen(argv[1],"r");
fscanf(f,"%d\n",&size);
size = 1 << size;
  if (size >= (1 << 20)){
    printf("Size (%u) is too large: size is limited to 2^20\n",size);
    exit(-1);
  }
vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
for (int i=0; i<size; i++){
    fscanf(f, "%u\n",&(vec[i]));
  }
unsigned int sum =0;

reduce(vec, &sum, size);
printf("sum=%u\n", sum);
return 0;

}

